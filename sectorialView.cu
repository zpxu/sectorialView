#include "hip/hip_runtime.h"
/// Definitions of wrapper functions
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "sectorialView_kernel.cuh"

//Round a / b to nearest higher integer value
uint cuda_iDivUp(uint a, uint b)
{
	return (a + (b - 1)) / b;
}

void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
	numThreads = min(blockSize, n);
	numBlocks = cuda_iDivUp(n, numThreads);
}

extern "C"
{

	void curandInit(hiprandState *state, int seed, uint numNodes)
	{
		uint numThreads, numBlocks;
		computeGridSize(numNodes, 512, numBlocks, numThreads);
		setup_RNG_kernel << <numBlocks, numThreads >> >(state, seed, numNodes);
		getLastCudaError("Kernel execution failed: setup_RNG_kernel");
	}

	void cudaGenerateUniform(float *result, hiprandState *state, uint numNodes)
	{
		uint numThreads, numBlocks;
		computeGridSize(numNodes, 512, numBlocks, numThreads);
		generate_uniform_kernel << <numBlocks, numThreads >> >(result, state, numNodes);
		getLastCudaError("Kernel execution failed: generate_uniform_kernel");
	}
} // extern "C"